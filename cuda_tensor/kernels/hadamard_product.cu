
#include <hip/hip_runtime.h>

extern "C" __global__ void
hadamard_product(const float *pX, const float *pY, float *pZ, int elements){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < elements) pZ[tid] = pX[tid] * pY[tid];
}
